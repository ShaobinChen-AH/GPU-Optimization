#include "hip/hip_runtime.h"
//最原始的代码，实现数组元素相加的功能，后续我们会从各个角度将此程序优化

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256
__global__ void reduce0(float *d_in, float * d_out) {
    __shared__ float sdata[THREAD_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = d_in[i];
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

bool check(float *out, float *res, int n) {
    for (int i = 0; i < n; i++) 
        if (out[i] != res[i]) return false;
    return true;
}

int main(){
    const int N = 32 * 1024 * 1024;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK;
    float *out = (float *)malloc((N / THREAD_PER_BLOCK) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (N / THREAD_PER_BLOCK) * sizeof(float));
    float *res = (float *)malloc((N / THREAD_PER_BLOCK) * sizeof(float));

    for (int i = 0; i < N; i++) {
        a[i] = 1;
    }

    for (int i = 0; i < block_num; i++) {
        float cur = 0;
        for (int j = 0; j < THREAD_PER_BLOCK; j++) cur += a[i * THREAD_PER_BLOCK + j];
        res[i] = cur;
    }

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(N / THREAD_PER_BLOCK, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    reduce0<<<grid, Block>>>(d_a, d_out);

    hipMemcpy(out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(out, res, block_num)) printf("the ans is right\n");
    else {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++) printf("%lf", out[i]);
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);
}
